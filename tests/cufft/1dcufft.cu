#define NX 3
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


int main(int argc, char** argv) {
    hipfftHandle plan;
    hipfftComplex *data;

    int mem_size = sizeof(hipfftComplex)*NX;
    hipMalloc((void**)&data, mem_size);

    float2 * h_signal = (float2*)malloc(NX*sizeof(float2));

    for (int i = 0; i < NX ; ++i) {
        h_signal[i].x = 1;
        h_signal[i].y = 0;
    }

    printf(">>> memsize = %d\n", mem_size);
    hipMemcpy(data, h_signal, mem_size, hipMemcpyHostToDevice);

    printf(">>> Create a 1D FFT plan.\n");
    hipfftPlan1d(&plan, NX, HIPFFT_C2C, 1);

    printf(">>> Use the CUFFT plan to transform the signal in place.\n");
    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    hipMemcpy(h_signal, data, mem_size, hipMemcpyHostToDevice);

    float2 * h_signal_fft;

    hipMemcpy(h_signal_fft, data, mem_size, hipMemcpyDeviceToHost);
    //printf("x = %f, y = %f\n", h_signal_fft[1].x, h_signal_fft[1].y);

    for (int i=0; i < NX; ++i){
        printf("h_signal_fft[%d] = %f + j%f\n", i, h_signal_fft[i].x, h_signal_fft[i].y); //h_signal_fft[i]);
    }

    //printf(">>> Inverse transform the signal in place.\n");
    //cufftExecC2C(plan, data, data, CUFFT_INVERSE);

    printf(">>> Note:\n");
    printf("(1) Divide by number of elements in data set to get back original data\n");
    printf("(2) Identical pointers to input and output arrays implies in-place\n");
    printf("    transformation\n");

    printf(">>> Destroy the CUFFT plan.\n");
    hipfftDestroy(plan);
    hipFree(data);
    return 0;
}
